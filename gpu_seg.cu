#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <deque>
#include <vector>

#include "gpu_seg.hpp"
#include "cpu_seg.hpp"
#include "cpu_utils.hpp"

/*
 * TODO:
 * - debug
 * - use reduction on excess flow to effectively limit the number of iterations
 */


///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
void __global__ gpu_init(int *global_e, int *to_source, int gridx, int gridy)
{
    int *e = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(global_e));
    int *src_edg_ptr = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(to_source));
    int src_edg;

    for (int i = 0; i < 8; i++) {
        src_edg = *src_edg_ptr;

        *src_edg_ptr = 2*src_edg;     // writeback
        *e = src_edg;                 // writeback

        e = NEXT_PTR(e);
        src_edg_ptr = NEXT_PTR(src_edg_ptr);
    }
}

void __global__ gpu_push_horiz_tiles(int *global_e, int *global_h, int *right_edges,
  int *left_edges, int gridx, int gridy)
{
        int current_h = global_h[GLOBAL_XY(threadIdx.x*32, threadIdx.y)];
        int current_e = global_e[GLOBAL_XY(threadIdx.x*32, threadIdx.y)];
        int next_h = global_h[GLOBAL_XY((threadIdx.x + 1)*32, threadIdx.y)];
        int next_e = global_e[GLOBAL_XY((threadIdx.x + 1)*32, threadIdx.y)];

        int right_edge = right_edges[GLOBAL_XY(threadIdx.x*32, threadIdx.y)];
        int left_edge = left_edges[GLOBAL_XY(threadIdx.x*32, threadIdx.y)];

        int delta = 0;
        if (current_h > next_h)
            delta = MIN(right_edge, current_e);
        else if (current_h < next_h)
            delta = -MIN(left_edge, next_e);
        right_edge  -= delta;
        current_e   -= delta;
        left_edge   += delta;
        next_e      += delta;

        if (threadIdx.x != gridx - 1) {
            right_edges[GLOBAL_XY(threadIdx.x*32, threadIdx.y)] = right_edge;
            left_edges[GLOBAL_XY(threadIdx.x*32, threadIdx.y)] = left_edge;
            global_e[GLOBAL_XY(threadIdx.x*32, threadIdx.y)] = current_e;
            global_e[GLOBAL_XY((threadIdx.x + 1)*32, threadIdx.y)] = next_e;
        }
}
void __global__ gpu_push_vertical_tiles(int *global_e, int *global_h, int *down_edges,
  int *up_edges, int gridx, int gridy)
{
        int current_h = global_h[GLOBAL_XY(threadIdx.x, threadIdx.y*32)];
        int current_e = global_e[GLOBAL_XY(threadIdx.x, threadIdx.y*32)];
        int next_h = global_h[GLOBAL_XY(threadIdx.x, (threadIdx.y + 1)*32)];
        int next_e = global_e[GLOBAL_XY(threadIdx.x, (threadIdx.y + 1)*32)];

        int down_edge = down_edges[GLOBAL_XY(threadIdx.x, threadIdx.y*32)];
        int up_edge = up_edges[GLOBAL_XY(threadIdx.x, threadIdx.y*32)];

        int delta = 0;
        if (current_h > next_h)
            delta = MIN(down_edge, current_e);
        else if (current_h < next_h)
            delta = -MIN(up_edge, next_e);
        down_edge  -= delta;
        current_e   -= delta;
        up_edge   += delta;
        next_e      += delta;

        if (threadIdx.y != gridy - 1) {
            down_edges[GLOBAL_XY(threadIdx.x, threadIdx.y*32)] = down_edge;
            up_edges[GLOBAL_XY(threadIdx.x, threadIdx.y*32)] = up_edge;
            global_e[GLOBAL_XY(threadIdx.x, threadIdx.y*32)] = current_e;
            global_e[GLOBAL_XY(threadIdx.x, (threadIdx.y + 1)*32)] = next_e;
        }
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
void __global__ gpu_push(int *global_e, int *global_h, int *right_edges,
  int *left_edges, int *up_edges, int *down_edges, int *to_source, int *to_sink, int gridx, int gridy)
{
    extern __shared__ int shared[];
    int *e = THREAD_DATA_OFFSET_PTR(&shared[0]);
    int *h = THREAD_DATA_OFFSET_PTR(&shared[32*32]);
    
    int *tmp_e = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(global_e));
    int *tmp_h = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(global_h));

    for (int i = 0; i < 8; i++) {
        *e = *tmp_e;
        *h = *tmp_h;
        e = NEXT_PTR(e); tmp_e = NEXT_PTR(tmp_e);
        h = NEXT_PTR(h); tmp_h = NEXT_PTR(tmp_h);
    }
    __syncthreads();


    int delta;
    int current_h, current_e;
    int next_h, next_e;
    int *right_edge_ptr, *left_edge_ptr;
    int right_edge, left_edge;

///////////////////////////////////////////////////////////////////////////////
    /* horizontal push */
    e = THREAD_DATA_OFFSET_PTR(&shared[0]);
    h = THREAD_DATA_OFFSET_PTR(&shared[32*32]);

    right_edge_ptr  = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(right_edges));
    left_edge_ptr   = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(left_edges));
    current_h = *h;
    current_e = *e;

    for (int i = 0; i < 7; i++) {
        next_h = *NEXT_PTR(h);
        next_e = *NEXT_PTR(e);

        right_edge = *right_edge_ptr;
        left_edge = *left_edge_ptr;

        if (current_h > next_h)
            delta = MIN(right_edge, current_e);
        else if (current_h < next_h)
            delta = -MIN(left_edge, next_e);
        else
            delta = 0;
        right_edge  -= delta;
        current_e   -= delta;
        left_edge   += delta;
        next_e      += delta;

        /* write back */
        *right_edge_ptr = right_edge;
        *left_edge_ptr = left_edge;
        *e = current_e;

        current_h = next_h;
        current_e = next_e;
        e = NEXT_PTR(e);
        h = NEXT_PTR(h);
        
        right_edge_ptr = NEXT_PTR(right_edge_ptr);
        left_edge_ptr = NEXT_PTR(left_edge_ptr);
    }
    
    __syncthreads();

    /* 8th iteration */
    if (threadIdx.y != BLOCK_HEIGHT - 1) {
        next_h = (&shared[32*32])[LOCAL_XY((threadIdx.y + 1)*8, threadIdx.x)];
        next_e = (&shared[0])[LOCAL_XY((threadIdx.y + 1)*8, threadIdx.x)];

        right_edge = *right_edge_ptr;
        left_edge = *left_edge_ptr;

        if (current_h > next_h)
            delta = MIN(right_edge, current_e);
        else if (current_h < next_h)
            delta = -MIN(left_edge, next_e);
        else
            delta = 0;
        right_edge  -= delta;
        current_e   -= delta;
        left_edge   += delta;
        next_e      += delta;
        
        /* write back */
        *right_edge_ptr = right_edge;
        *left_edge_ptr = left_edge;
        *e = current_e;
        (&shared[0])[LOCAL_XY((threadIdx.y + 1)*8, threadIdx.x)] = next_e;

    } 
    // sync delayed... (a)

///////////////////////////////////////////////////////////////////////////////
    /*
     * vertical push
     * NOTE: Unfortunately, I have found no way to avoid bank conflicts in the vertical push. Too bad :(
     */
    /* edges are read normally, it changes only for e and h */
    e = VTHREAD_DATA_OFFSET_PTR(&shared[0]);
    h = VTHREAD_DATA_OFFSET_PTR(&shared[32*32]);

    right_edge_ptr = VTHREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(down_edges));
    left_edge_ptr =  VTHREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(up_edges));

    __syncthreads(); // here! (a)

    current_h = *h;
    current_e = *e;

    for (int i = 0; i < 7; i++) {
        next_h = *VNEXT_PTR(h);
        next_e = *VNEXT_PTR(e);

        right_edge = *right_edge_ptr;
        left_edge = *left_edge_ptr;

        if (current_h > next_h)
            delta = MIN(right_edge, current_e);
        else if (current_h < next_h)
            delta = -MIN(left_edge, next_e);
        else
            delta = 0;
        right_edge  -= delta;
        current_e   -= delta;
        left_edge   += delta;
        next_e      += delta;
        
        /* write back */
        *right_edge_ptr = right_edge;
        *left_edge_ptr = left_edge;
        *e = current_e;

        current_h = next_h;
        current_e = next_e;
        e = VNEXT_PTR(e);
        h = VNEXT_PTR(h);
        
        right_edge_ptr = VNEXT_PTR(right_edge_ptr);
        left_edge_ptr = VNEXT_PTR(left_edge_ptr);
    }
    
    __syncthreads();

    /* 8th iteration */
    if (threadIdx.y != BLOCK_HEIGHT - 1) {
        next_h = *VNEXT_PTR(h);
        next_e = *VNEXT_PTR(e);

        right_edge = *right_edge_ptr;
        left_edge = *left_edge_ptr;

        if (current_h > next_h)
            delta = MIN(right_edge, current_e);
        else if (current_h < next_h)
            delta = -MIN(left_edge, next_e);
        else
            delta = 0;
        right_edge  -= delta;
        current_e   -= delta;
        left_edge   += delta;
        next_e      += delta;
        
        /* write back */
        *right_edge_ptr = right_edge;
        *left_edge_ptr = left_edge;
        *e = current_e;
        *VNEXT_PTR(e) = next_e;
    }
    __syncthreads();

    
///////////////////////////////////////////////////////////////////////////////
    /* source push */
    e = THREAD_DATA_OFFSET_PTR(&shared[0]);
    h = THREAD_DATA_OFFSET_PTR(&shared[32*32]);

    right_edge_ptr = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(to_source));

    for (int i = 0; i < 8; i++) {
        current_h = *h;
        current_e = *e;

        right_edge = *right_edge_ptr;

        if (current_h > 32*32*gridx*gridy)
            delta = MIN(right_edge, current_e);
        else
            delta = 0;
        right_edge  -= delta;
        current_e   -= delta;
        
        /* write back */
        *right_edge_ptr = right_edge;
        *e = current_e;

        e = NEXT_PTR(e);
        h = NEXT_PTR(h);
        right_edge_ptr = NEXT_PTR(right_edge_ptr);
    }

    /* NO NEED TO SYNC HERE */

///////////////////////////////////////////////////////////////////////////////
    /* sink push */
    e = THREAD_DATA_OFFSET_PTR(&shared[0]);
    h = THREAD_DATA_OFFSET_PTR(&shared[32*32]);

    right_edge_ptr = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(to_sink));

    for (int i = 0; i < 8; i++) {
        current_h = *h;
        current_e = *e;

        right_edge = *right_edge_ptr;

        if (current_h > 0)
            delta = MIN(right_edge, current_e);
        else
            delta = 0;
        right_edge  -= delta;
        current_e   -= delta;
        
        /* write back */
        *right_edge_ptr = right_edge;
        *e = current_e;

        e = NEXT_PTR(e);
        h = NEXT_PTR(h);
        right_edge_ptr = NEXT_PTR(right_edge_ptr);
    }

    __syncthreads();
    
///////////////////////////////////////////////////////////////////////////////
    /* write back */
    e = THREAD_DATA_OFFSET_PTR(&shared[0]);
    tmp_e = THREAD_DATA_OFFSET_PTR(BLOCK_DATA_START_PTR(global_e));
    for (int i = 0; i < 8; i++) {
        *tmp_e = *e;
        e = NEXT_PTR(e);
        tmp_e = NEXT_PTR(tmp_e);
    }
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
/* run this 32x32 */
void __global__ gpu_relabel(int *global_e, int *global_h, int *right_edges,
  int *left_edges, int *up_edges, int *down_edges, int *to_source, int *to_sink, int gridx, int gridy)
{
    int x = blockIdx.x*32 + threadIdx.x;
    int y = blockIdx.y*32 + threadIdx.y;

    int h = global_h[GLOBAL_XY(x, y)];

    int down = (y < 32*gridy - 1) ? down_edges[GLOBAL_XY(x, y)] : 0;
    int left = (x > 0) ? left_edges[GLOBAL_XY(x, y)] : 0;
    int right = (x < 32*gridx - 1) ? right_edges[GLOBAL_XY(x, y)] : 0;
    int up = (y > 0) ? up_edges[GLOBAL_XY(x, y)] : 0;
    int sink = to_sink[GLOBAL_XY(x, y)];
    int source = to_source[GLOBAL_XY(x, y)];

    int min_height = INF;
    int relabel = global_e[GLOBAL_XY(x, y)] > 0;

    if (down > 0) {
        int hdown = global_h[GLOBAL_XY(x, y+1)];
        if (h > hdown)
            relabel = 0;
        min_height = MIN(min_height, hdown);
    }
    if (up > 0) {
        int hup = global_h[GLOBAL_XY(x, y-1)];
        if (h > hup)
            relabel = 0;
        min_height = MIN(min_height, hup);
    }
    if (left > 0) {
        int hleft = global_h[GLOBAL_XY(x - 1, y)];
        if (h > hleft)
            relabel = 0;
        min_height = MIN(min_height, hleft);
    }
    if (right > 0) {
        int hright = global_h[GLOBAL_XY(x+1, y)];
        if (h > hright)
            relabel = 0;
        min_height = MIN(min_height, hright);
    }
    if (source > 0) {
        if (h > 32*32*gridx*gridy)
            relabel = 0;
        min_height = MIN(min_height, 32*32*gridx*gridy);
    }
    if (sink > 0) {
        if (h > 0)
            relabel = 0;
        min_height = MIN(min_height, 0);
    }

    __syncthreads();

    if (relabel && min_height != INF)
        global_h[GLOBAL_XY(x, y)] = min_height + 1;
}


/* Image resolution is a multiplication of 32 */
void segmentation_gpu(int width, int height, const pixel_t *image,
  const pixel_t *marks, pixel_t *segmented_image)
{
    typedef ImageGraph::regular_node_t regular_node_t;
    typedef ImageGraph::node_t node_t;

    ImageGraph g(width, height);
    Histogram hist(width, height, image, marks);

    std::vector<bool> visited(height*width, false);
    std::deque<int> Q;



    /* REG_NEIGBHOURS, unfortunately, has to be 4 here */
    const int delt[ImageGraph::REG_NEIGHBOURS][2] = {
                {-1,  0},
      { 0, -1},           { 0, +1},
                {+1,  0}};

    /* neighbour edges */
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            regular_node_t &v = g.get(x, y);
            for (int i = 0; i < g.REG_NEIGHBOURS; i++) {
                if (in_range(y + delt[i][0], 0, height - 1) &&
                  in_range(x + delt[i][1], 0, width - 1)) {
                    const int dy = y + delt[i][0];
                    const int dx = x + delt[i][1];
                    regular_node_t &u = g.get(dx, dy);
                    v.c[i] = compute_edge(image[y*width + x], image[dy*width + dx]);
                }
            }
        }
    }

    /* source and sink edges */
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            regular_node_t &v = g.get(x, y);

            int k = 0;
            for (int i = 0; i < g.REG_NEIGHBOURS; i++)
                k = (k < v.c[i]) ? v.c[i] : k;
            k = k + 1;

            int i = y*width + x;
            if (marks[y*width + x] == WHITE) {
                v.c[g.SOURCE] = g.source.c[i] = k;
                v.c[g.SINK] = g.sink.c[i] = 0;
            } else if (marks[y*width + x] == BLACK) {
                v.c[g.SINK] = g.sink.c[i] = k;
                v.c[g.SOURCE] = g.source.c[i] = 0;
            } else {
                v.c[g.SOURCE] = MULT*LAMBDA*(-log(hist.prob_bg(image[i])));
                v.c[g.SINK] = MULT*LAMBDA*(-log(hist.prob_obj(image[i])));
                g.source.c[i] = v.c[g.SOURCE];
                g.sink.c[i] = v.c[g.SINK];
            }
        }
    }


    int *to_sink, *to_source, *up_e, *down_e, *right_e, *left_e;
    int *e, *h;
    to_sink = new int[width*height];
    to_source = new int[width*height];
    up_e = new int[width*height];
    down_e = new int[width*height];
    right_e = new int[width*height];
    left_e = new int[width*height];
    e = new int[width*height];
    h = new int[width*height];

    int gridx, gridy;

    gridx = width/32;
    gridy = height/32;
    for (unsigned x = 0; x < width; x++) {
        for (unsigned y = 0; y < height; y++) {
            to_sink[GLOBAL_XY(x, y)] = g.get(x, y).c[g.SINK];
            to_source[GLOBAL_XY(x, y)] = g.get(x, y).c[g.SOURCE];
            if (y != height - 1)
                up_e[GLOBAL_XY(x, y)] = g.get(x, y + 1).c[0];
            else
                up_e[GLOBAL_XY(x, y)] = 0;

            if (x != width - 1)
                left_e[GLOBAL_XY(x, y)] = g.get(x + 1, y).c[1];
            else
                left_e[GLOBAL_XY(x, y)] = 0;
            
            right_e[GLOBAL_XY(x, y)] = g.get(x, y).c[2];
            down_e[GLOBAL_XY(x, y)] = g.get(x, y).c[3];
            e[GLOBAL_XY(x, y)] = 0; 
            h[GLOBAL_XY(x, y)] = 0;
        }
    }

    int *cto_sink, *cto_source, *cup_e, *cdown_e, *cright_e, *cleft_e;
    int *ce, *ch;

    hipMalloc(&cto_sink, width*height*sizeof(int));
    hipMalloc(&cto_source, width*height*sizeof(int));
    hipMalloc(&cup_e, width*height*sizeof(int));
    hipMalloc(&cdown_e, width*height*sizeof(int));
    hipMalloc(&cright_e, width*height*sizeof(int));
    hipMalloc(&cleft_e, width*height*sizeof(int));
    hipMalloc(&ce, width*height*sizeof(int));
    hipMalloc(&ch, width*height*sizeof(int));

    hipMemcpy(cto_sink, to_sink, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cto_source, to_source, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cup_e, up_e, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cdown_e, down_e, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cright_e, right_e, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cleft_e, left_e, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ce, e, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ch, h, width*height*sizeof(int), hipMemcpyHostToDevice);

    
    dim3 relabel_dim(32, 32);
    dim3 other_dim(32, 4);
    dim3 grid_dim(gridx, gridy);
    hipError_t err;

    gpu_init<<<grid_dim, other_dim>>> (ce, cto_source, gridx, gridy);
    if (hipSuccess != (err = hipGetLastError())) {
        printf("#0 Error: %s\n", hipGetErrorString(err));
        goto cleanup;
    }
    

    // TODO: change this O(w*h) loop into while(reduction())
    for (int i = 0; i < width*height; i++) {
        gpu_push<<<grid_dim, other_dim, 32*32*8>>>(ce, ch, cright_e, cleft_e,
          cup_e, cdown_e, cto_source, cto_sink, gridx, gridy);
        if (hipSuccess != (err = hipGetLastError())) {
            printf("#1 Error: %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        gpu_push_horiz_tiles<<<dim3(gridx, height), 1>>>(ce, ch, cright_e,
          cleft_e, gridx, gridy);
        if (hipSuccess != (err = hipGetLastError())) {
            printf("#2 Error: %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        gpu_push_vertical_tiles<<<dim3(width, gridy), 1>>>(ce, ch, cdown_e,
          cup_e, gridx, gridy);
        if (hipSuccess != (err = hipGetLastError())) {
            printf("#3 Error: %s\n", hipGetErrorString(err));
            goto cleanup;
        }
        gpu_relabel<<<grid_dim, relabel_dim, 32*32*8>>>(ce, ch, cright_e,
          cleft_e, cup_e, cdown_e, cto_source, cto_sink, gridx, gridy);
        if (hipSuccess != (err = hipGetLastError())) {
            printf("#4 Error: %s\n", hipGetErrorString(err));
            goto cleanup;
        }
    }


    hipMemcpy(to_sink, cto_sink, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(to_source, cto_source, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(up_e, cup_e, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(down_e, cdown_e, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(right_e, cright_e, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(left_e, cleft_e, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(e, ce, width*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h, ch, width*height*sizeof(int), hipMemcpyDeviceToHost);


    /* Copying back */
    for (unsigned x = 0; x < width; x++) {
        for (unsigned y = 0; y < height; y++) {
            g.get(x, y).c[g.SINK] = to_sink[GLOBAL_XY(x, y)];
            g.get(x, y).c[g.SOURCE] = to_source[GLOBAL_XY(x, y)];

            if (y > 0)
                g.get(x, y).c[0] = up_e[GLOBAL_XY(x, y - 1)];
            else
                g.get(x, y).c[0] = 0;
            
            if (x > 0)
                g.get(x, y).c[1] = left_e[GLOBAL_XY(x - 1, y)];
            else
                g.get(x, y).c[1] = 0;

            g.get(x, y).c[2] = right_e[GLOBAL_XY(x, y)];
            g.get(x, y).c[3] = down_e[GLOBAL_XY(x, y)];

            g.get(x, y).overflow = e[GLOBAL_XY(x, y)];
            g.get(x, y).height = h[GLOBAL_XY(x, y)];
            if (g.get(x, y).height > 0)
                printf("%d \t%d \t| overflow = %d \t| height = %d\n", x, y, g.get(x, y).overflow, g.get(x, y).height); // DEBUG
        }
    }
    /* Make the image white */
    for (int i = 0; i < height*width; i++) {
        segmented_image[i].r = 255;
        segmented_image[i].g = 255;
        segmented_image[i].b = 255;
    }

    for (unsigned i = 0; i < g.width*g.height; i++) {
        if (g.source.c[i] > 0) {
            segmented_image[i].r = image[i].r;
            segmented_image[i].g = image[i].g;
            segmented_image[i].b = image[i].b;
            Q.push_back(i);
            visited[i] = true;
        }
    }

    while (!Q.empty()) {
        int vpos = Q.front();
        Q.pop_front();

        const int x = vpos % width;
        const int y = vpos / width;
        regular_node_t &v = g.nodes[vpos];
        for (int i = 0; i < g.REG_NEIGHBOURS; i++) {
            if (in_range(y + delt[i][0], 0, height - 1) &&
              in_range(x + delt[i][1], 0, width - 1)) {

                if (v.c[i] <= 0)
                    continue;

                const int dy = y + delt[i][0];
                const int dx = x + delt[i][1];
                const int upos = dy*width + dx;
                if (visited[upos])
                    continue;

                segmented_image[upos].r = image[upos].r;
                segmented_image[upos].g = image[upos].g;
                segmented_image[upos].b = image[upos].b;
                Q.push_back(upos);
                visited[upos] = true;
            }
        }
    }

cleanup:
    hipFree(cdown_e);
    hipFree(ce);
    hipFree(ch);
    hipFree(cleft_e);
    hipFree(cright_e);
    hipFree(cto_sink);
    hipFree(cto_source);
    hipFree(cup_e);

    delete [] down_e;
    delete [] e;
    delete [] h;
    delete [] left_e;
    delete [] right_e;
    delete [] to_sink;
    delete [] to_source;
    delete [] up_e;
}
